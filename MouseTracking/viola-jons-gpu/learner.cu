#include "learner.h"
#include <iostream>
using namespace std;
#include "const.h"
#include "utils.h"
#include <vector>
#include <hip/hip_runtime.h>
__host__ __device__ int Learner::predict(int *&X)
{
    return this->polarity * ((X[this->feature_index] >= this->threshold) ? 1 : -1);
}
__host__ __device__ int Learner::predict(int **&X, int size, double devide)
{
    int index = this->feature_index;
#ifdef __CUDA_ARCH__
    // On the device, use the device pointer
    return this->polarity * ((haar_feature_scaling(X, size, d_features_info[index].feature_type, d_features_info[index].i, d_features_info[index].j, d_features_info[index].w, d_features_info[index].h) / devide) >= this->threshold ? 1 : -1);
#else
    // On the host, use the host pointer
    return this->polarity * ((haar_feature_scaling(X, size, features_info[index].feature_type, features_info[index].i, features_info[index].j, features_info[index].w, features_info[index].h) / devide) >= this->threshold ? 1 : -1);
#endif
}
